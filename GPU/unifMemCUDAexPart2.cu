
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define SZ 8

__global__ void AplusB(int *ret, int a, int b) {
  ret[threadIdx.x] = a + b + threadIdx.x;
}

int main() {
  int *ret;
  hipMallocManaged(&ret, SZ * sizeof(int));
  AplusB<<<1, SZ>>>(ret, 10, 100);
  hipDeviceSynchronize();
  for (int i = 0; i < SZ; i++)
    printf("%d: A+B = %d\n", i, ret[i]);
  hipFree(ret);
  return 0;
}
