#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <iostream>

#define RADIUS 3
#define N 1000000

void initializeWeights(float* weights) {
    weights[0] = 0.05f;
    weights[1] = 0.10f;
    weights[2] = 0.20f;
    weights[3] = 0.30f;
    weights[4] = 0.20f;
    weights[5] = 0.10f;
    weights[6] = 0.05f;
}

void initializeArray(float* in) {
    for (int i = 0; i < N; i++) {
        in[i] = rand() % 10;
    }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float* weights, float* in, float* out) {
    int i = sIdx + blockIdx.x * blockDim.x + threadIdx.x;
    if (i < eIdx) {
        out[i] = 0;
        //loop over all elements in the stencil
        for (int j = -RADIUS; j <= RADIUS; j++) {
            out[i] += weights[j + RADIUS] * in[i + j];
        }
        out[i] = out[i] / (2 * RADIUS + 1);
    }
}

int main() {
    int wsize = 2 * RADIUS + 1;
    //allocate resources
    float* weights = new float[wsize];
    float* in = new float[N];
    float* out = new float[N];
    initializeWeights(weights);
    initializeArray(in);

    float* d_weights;
    hipMalloc(&d_weights, wsize * sizeof(float));
    
    float* d_in;
    hipMalloc(&d_in, N * sizeof(float));
    
    float* d_out;
    hipMalloc(&d_out, N * sizeof(float));

    hipMemcpy(d_weights, weights, wsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_in, in, N*sizeof(float), hipMemcpyHostToDevice);

    applyStencil1D <<<(N+511) / 512, 512 >>> (RADIUS, N - RADIUS, d_weights, d_in, d_out);

    hipMemcpy(out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);

    //free resources
    delete[] weights;
    delete[] in;
    delete[] out;

    hipFree(d_weights);
    hipFree(d_in);
    hipFree(d_out);
}
