#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SZ 8
__global__ void write(int *ret, int a, int b) {
    ret[threadIdx.x] = a + b + threadIdx.x;
}
__global__ void append(int *ret, int a, int b) {
    ret[threadIdx.x] += a + b + threadIdx.x;
}

int main() {
    int *ret;
    hipMallocManaged(&ret, SZ * sizeof(int));

    // set direct access hint
    hipMemAdvise(ret, SZ * sizeof(int), hipMemAdviseSetAccessedBy, hipCpuDeviceId);  

    // pages populated in GPU memory
    write<<< 1, SZ >>>(ret, 10, 100);            
    hipDeviceSynchronize();

    // print operation - directManagedMemAccessFromHost=1: CPU accesses GPU memory directly without migrations
    // If directManagedMemAccessFromHost was 0, then CPU faults and triggers device-to-host migration
    for (int i = 0; i < SZ; i++)
        printf("%d: A+B = %d\n", i, ret[i]);        
                                                    
    // directManagedMemAccessFromHost=1: GPU accesses GPU memory without migrations
    // If directManagedMemAccessFromHost was 0, then CPU faults and triggers device-to-host migration
    append <<<1, SZ>>>(ret, 10, 100);            
    hipDeviceSynchronize(); 
    printf("\nNew results:\n");
    for (int i = 0; i < SZ; i++)
        printf("%d: A+B = %d\n", i, ret[i]);
    hipFree(ret);
    return 0;
}
