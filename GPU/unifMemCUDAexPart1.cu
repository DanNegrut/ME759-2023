
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define SZ 8

__global__ void AplusB(int *ret, int a, int b) {
  ret[threadIdx.x] = a + b + threadIdx.x;
}

int main() {
  int *ret;
  hipMalloc(&ret, SZ * sizeof(int));
  AplusB<<<1, SZ>>>(ret, 10, 100);
  int *host_ret = (int *)malloc(SZ * sizeof(int));
  hipMemcpy(host_ret, ret, SZ * sizeof(int), hipMemcpyDefault);
  for (int i = 0; i < SZ; i++)
    printf("%d: A+B = %d\n", i, host_ret[i]);
  free(host_ret);
  hipFree(ret);
  return 0;
}
